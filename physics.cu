#include "hip/hip_runtime.h"
/*
Compile with: 

    gcc physics.c -o physics -O2 -lm -std=c99 

    -O2      Optimization
    -lm      Link to math lib
    -std=c99 Use of for(;;;) with declaration among other things

Usage (3D viewer):

    ./physics > data && ./physicsViewer data
    
Usage (debug):

    ./physics

Jan Mas Rovira
Andrés Mingorance López
Albert Puente Encinas
*/

#include <stdio.h>  // e.g. printf
#include <stdlib.h> // e.g. malloc, RAND_MAX, exit
#include <math.h>   // e.g. sin, abs
#include <sys/time.h>
#include <hip/hip_runtime.h>

// Algorithm parameters
#define N 256*4
#define ITERATIONS 2000
#define G 9.81
#define BOUNCE_DECAY 0.5
#define GLOBAL_DECAY 0.004
#define POINT_RADIUS 0.3
#define TIME_SPEED 0.0075
#define MAX_TRIES 1e4
#define SEED 27

#define DUMP_RATIO 4

// CUDA Variables
unsigned int nThreads = 1024;
unsigned int nBlocks = N/nThreads;  // N multiple de nThreads

// c++ style
#define bool int
#define true 1
#define false 0

#define WALLS true

// Timers
unsigned long long initialGenTime;
unsigned long long interactionsTime;
unsigned long long worldInteractionsTime;
unsigned long long gravityTime;
unsigned long long advanceTime;
unsigned long long frameTime;
unsigned long long totalTime;

inline void tic(unsigned long long* time) {
    struct timeval t;
    gettimeofday(&t, NULL);
    *time = t.tv_sec*1000000 + t.tv_usec - *time;
    
}
#define toc tic

// Output toggles
bool DUMP;

typedef struct {
    float x, y, z;
} Vector;

typedef struct {
    float x, y, z;      // Position
    Vector velocity;    // Velocity
} Point;

typedef struct {
    Point points[N];
} PointSet;

void checkCudaError(char msg[]) {
    hipError_t error;
    error = hipGetLastError();
    if (error) {
        printf("Error: %s: %s\n", msg, hipGetErrorString(error));
        exit(1);
    }
}

inline float dist(Point* a, Point* b) {
    return sqrt(pow(a->x - b->x, 2)+pow(a->y - b->y, 2)+pow(a->z - b->z, 2));
}

__device__ inline float gpu_dist(Point* a, Point* b) {
    return sqrt(pow(a->x - b->x, 2)+pow(a->y - b->y, 2)+pow(a->z - b->z, 2));
}

__device__ inline float distNext(Point* a, Point* b) {
    return sqrt( pow(a->x + a->velocity.x*TIME_SPEED - (b->x + b->velocity.x*TIME_SPEED), 2)+
                 pow(a->y + a->velocity.y*TIME_SPEED - (b->y + b->velocity.y*TIME_SPEED), 2)+
                 pow(a->z + a->velocity.z*TIME_SPEED - (b->z + b->velocity.z*TIME_SPEED), 2));
}

bool collides(Point* p, PointSet* PS, int from, int to) {
    for (int i = from; i < to; ++i) {
        if (dist(p, &PS->points[i]) < POINT_RADIUS*2) {
            return true;
        }
    }
    return false;
}

__device__ Vector diffVector(Point* a, Point* b) {
    Vector v;
    float e = 1e-40;
    v.x = a->x - b->x;
    if (abs(v.x) < e) v.x = 0;
    v.y = a->y - b->y;
    if (abs(v.y) < e) v.y = 0;
    v.z = a->z - b->z;
    if (abs(v.z) < e) v.z = 0;
    return v;
}

__device__ inline float dotProduct(Vector a, Vector b) {
    return a.x*b.x + a.y*b.y + a.z*b.z;
}


__global__ void kernel_interaction(PointSet* P, PointSet* Q) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    for (int j = i + 1; j < N; ++j) {
        if (i == j) return;
        
        Point* a = &P->points[i];
        Point* b = &P->points[j];
        
        float distance = gpu_dist(a, b);
        if (distance > 2*POINT_RADIUS + 0.05) return;
        
        if (distance == 0) return; // AVOID NAN, PROVISIONAL
        
        if (distance < distNext(a, b)) return;
        
        // http://stackoverflow.com/questions/345838/ball-to-ball-collision-detection-and-handling
        
        // Get the components of the velocity vectors which are parallel to the collision.
        // The perpendicular component remains the same for both fish
        Vector collision = diffVector(a, b);
        
        //
        //distance = 2*POINT_RADIUS;
        collision.x /= distance;
        collision.y /= distance;
        collision.z /= distance;
        
        float aci = dotProduct(collision, a->velocity); 
        float bci = dotProduct(collision, b->velocity); 

        // Solve for the new velocities using the 1-dimensional elastic collision equations.
        // Turns out it's really simple when the masses are the same.
        float acf = bci;
        float bcf = aci;

        // Replace the collision velocity components with the new ones
        Point* aq = &Q->points[i];
        Point* bq = &Q->points[j];
        aq->velocity.x += (acf - aci) * collision.x;
        aq->velocity.y += (acf - aci) * collision.y;
        aq->velocity.z += (acf - aci) * collision.z;
        
        bq->velocity.x += (bcf - bci) * collision.x;
        bq->velocity.y += (bcf - bci) * collision.y;
        bq->velocity.z += (bcf - bci) * collision.z;
    }
}

void computeInteraction(PointSet* gpu_P, PointSet* gpu_Q) {
    kernel_interaction<<<nBlocks, nThreads>>>(gpu_P, gpu_Q);
    checkCudaError((char *) "kernel call in interaction");    
    hipDeviceSynchronize();
}

__global__ void kernel_gravity(PointSet* P) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    P->points[id].velocity.y -= G*TIME_SPEED;    
}

void applyGravity(PointSet* gpu_P) {
    kernel_gravity<<<nBlocks, nThreads>>>(gpu_P);
    checkCudaError((char *) "kernel call in applyGravity");    
    hipDeviceSynchronize();
}

__global__ void kernel_advance(PointSet* P, PointSet* Q) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    Point* p = &P->points[id];
    p->x += p->velocity.x*TIME_SPEED;
    p->y += p->velocity.y*TIME_SPEED;
    p->z += p->velocity.z*TIME_SPEED;
    p->velocity.x *= (1-GLOBAL_DECAY);
    p->velocity.y *= (1-GLOBAL_DECAY);
    p->velocity.z *= (1-GLOBAL_DECAY);
    
    Q->points[id] = *p;
}

void advanceAndCopy(PointSet* gpu_P, PointSet* gpu_Q) {
    kernel_advance<<<nBlocks, nThreads>>>(gpu_P, gpu_Q);
    checkCudaError((char *) "kernel call in advance");    
    hipDeviceSynchronize();
}

__device__ inline void ifelse(bool condition, float* dest, float a, float b) {
    *dest = condition*a + !condition*b;    
}

__global__ void kernel_world(PointSet* P) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    
    Point* p = &P->points[id];
    
    ifelse(p->y < POINT_RADIUS, &p->y, p->y, POINT_RADIUS);
    
    ifelse(p->y < POINT_RADIUS, &p->velocity.y, 
           p->velocity.y, abs(p->velocity.y)*(1.0 - BOUNCE_DECAY));

    if (WALLS) { // 4 walls x = -10, 10 and z = -10, 10
        if (p->x < -10.0 + POINT_RADIUS) {
            p->x = -10 + POINT_RADIUS;
            p->velocity.x = abs(p->velocity.x) * (1.0 - BOUNCE_DECAY);
        }
        else if (p->x > 10.0 - POINT_RADIUS) {
            p->x = 10 - POINT_RADIUS;
            p->velocity.x = -abs(p->velocity.x) * (1.0 - BOUNCE_DECAY);
        }            
        
        if (p->z < -10.0 + POINT_RADIUS) {
            p->z = -10 + POINT_RADIUS;
            p->velocity.z = abs(p->velocity.z) * (1.0 - BOUNCE_DECAY);
        }            
        else if (p->z > 10.0 - POINT_RADIUS) {
            p->z = 10 - POINT_RADIUS;
            p->velocity.z = -abs(p->velocity.z) * (1.0 - BOUNCE_DECAY);
        }        
    } 
}

void computeInteractionWorld(PointSet* gpu_P) {
    kernel_world<<<nBlocks, nThreads>>>(gpu_P);
    checkCudaError((char *) "kernel call in interactionWorld");    
    hipDeviceSynchronize();
}


void computePhysics(PointSet* gpu_P, PointSet* gpu_Q) {
    tic(&gravityTime);
    applyGravity(gpu_Q); 
    toc(&gravityTime);
    
    tic(&worldInteractionsTime);
    computeInteractionWorld(gpu_Q);
    tic(&worldInteractionsTime);
    
    tic(&interactionsTime);
    computeInteraction(gpu_P, gpu_Q);
    tic(&interactionsTime);
    
    tic(&advanceTime);
    advanceAndCopy(gpu_Q, gpu_P);
    toc(&advanceTime);
}

void generateInitialConfiguration(PointSet* gpu_P, PointSet* gpu_Q) {
    tic(&initialGenTime);
    PointSet* P = (PointSet*) malloc(sizeof(PointSet));
    
    for (int i = 0; i < N; ++i) {
        Point* p = &P->points[i]; 

        p->x = 10.0*(float)rand()/(float)(RAND_MAX) - 5.0;
        p->y = 30.0*(float)rand()/(float)(RAND_MAX) + 1.0;
        p->z = 10.0*(float)rand()/(float)(RAND_MAX) - 5.0;       
        
        p->velocity.x = 0.0;
        p->velocity.y = -3.5;
        p->velocity.z = 0.0;
                
        int tests = 0;
        while (tests < MAX_TRIES && collides(p, P, 0, i)) {

            p->x = 10.0*(float)rand()/(float)(RAND_MAX) - 5.0;
            p->y = 30.0*(float)rand()/(float)(RAND_MAX) + 1.0;
            p->z = 10.0*(float)rand()/(float)(RAND_MAX) - 5.0;
            ++tests;
        }
        if (tests == MAX_TRIES) {
            printf("Error during the generation of the initial conf.\n");
            exit(1);
        }
    }
    
    hipMemcpy(gpu_P, P, sizeof(PointSet), hipMemcpyHostToDevice);
    checkCudaError((char *) "host -> gpu_P");
    
    hipMemcpy(gpu_Q, P, sizeof(PointSet), hipMemcpyHostToDevice);
    checkCudaError((char *) "host -> gpu_Q");
    
    hipDeviceSynchronize();
    
    toc(&initialGenTime);
}

void DUMPInitialParams() {
    printf("%i %i\n", N, ITERATIONS);
}

__global__ void kernel_print(PointSet* P) {
     int id = blockIdx.x * blockDim.x + threadIdx.x;
     printf("%f %f %f\n", P->points[id].x, P->points[id].y, P->points[id].z);
}

void dump(PointSet* gpu_P) {
    kernel_print<<<nBlocks, nThreads>>>(gpu_P);
    checkCudaError((char *) "kernel call in interaction");    
    hipDeviceSynchronize();
}

void initTimes() {
    initialGenTime = 0;
    interactionsTime = 0;
    worldInteractionsTime = 0;
    gravityTime = 0;
    advanceTime = 0;
    frameTime = 0;
    totalTime = 0;
}

void printTimes() {
    printf("Sequential physics algorithm has finished:\n");
    printf("    Init gen:     %f s.\n", (double)initialGenTime/1000000);
    printf("    Interactions: %f s.\n", (double)interactionsTime/1000000);
    printf("    World int.:   %f s.\n", (double)worldInteractionsTime/1000000);
    printf("    Gravity:      %f s.\n", (double)gravityTime/1000000);
    printf("    Advance:      %f s.\n", (double)advanceTime/1000000);
    // printf("    Avg. frame:   %f s.\n", (double)frameTime/(ITERATIONS*1000000));
    printf("    Total time:   %f s.\n", (double)totalTime/1000000);
}

void sequentialPhysics() {
    
    DUMPInitialParams();  
        
    PointSet* gpu_P;
    PointSet* gpu_Q;
    hipMalloc((void **) &gpu_P, sizeof(PointSet));
    checkCudaError((char *) "hipMalloc of P");
    hipMalloc((void **) &gpu_Q, sizeof(PointSet));
    checkCudaError((char *) "hipMalloc of Q");
    
    tic(&totalTime);
    srand(SEED);
    generateInitialConfiguration(gpu_P, gpu_Q); // *CPU_P = *gpu_P = *gpu_Q
    
    
    for (int i = 0; i < ITERATIONS; ++i) {
        tic(&frameTime);        
        computePhysics(gpu_P, gpu_Q);      
        if (DUMP) {
            if (i%DUMP_RATIO == 0) dump(P);
        }
        else printf("IT %i\n", i);
        
        toc(&frameTime);    
    }
    
    toc(&totalTime);
    if (!DUMP) printTimes();   
}

int main(int argc, char** argv) {
    DUMP = (argc == 1);
    sequentialPhysics();
    return 0;
}

